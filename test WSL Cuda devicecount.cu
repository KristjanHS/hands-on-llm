#include <hip/hip_runtime.h>
#include <iostream>
int main() {
  int count;
  hipError_t err = hipGetDeviceCount(&count);
  if (err != hipSuccess) {
    std::cerr << "CUDA error: " << hipGetErrorString(err) << "\n";
    return 1;
  }
  std::cout << "Number of CUDA devices: " << count << "\n";
}
